#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cufftMp.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <complex.h>
#include "hip/hip_complex.h"
#include "proto.h"
#include "errcodes.h"
#include <time.h>
#include <unistd.h>

#if defined(CUFFTMP) && defined(USE_FFTW)



__global__ void write_grid(
	int num_w_planes,
	int xaxis,
	int yaxis,
	hipfftDoubleComplex * fftwgrid,
	double * grid,
	int iw)
{
  unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;
  if(gid<yaxis*xaxis)
    {
      unsigned int fftwindex2D = gid;
      unsigned int fftwindex = 2*(fftwindex2D + iw * xaxis * yaxis);
      fftwgrid[fftwindex2D].x = grid[fftwindex];
      fftwgrid[fftwindex2D].y = grid[fftwindex+1];
    }  
}


__global__ void write_gridss(
			     int num_w_planes,
			     int xaxis,
			     int yaxis,
			     hipfftDoubleComplex * fftwgrid,
			     double * gridss,
			     double norm,
			     int iw)
  
{
  unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;
  if(gid<yaxis*xaxis)
    {
      unsigned int fftwindex2D = gid;
      unsigned int fftwindex = 2*(fftwindex2D + iw * xaxis * yaxis);
      gridss[fftwindex] = norm*fftwgrid[fftwindex2D].x;
      gridss[fftwindex+1] = norm*fftwgrid[fftwindex2D].y;
    }
}







void cuda_fft(
	      int num_w_planes,
	      int grid_size_x,
	      int grid_size_y,
	      int xaxis,
	      int yaxis,
	      double * grid,
	      double * gridss,
	      int rank,
	      MPI_Comm comm)
{

  int ndevices;
  hipGetDeviceCount(&ndevices);
  hipSetDevice(rank % ndevices);

  if ( rank == 0 ) {
    if (0 == ndevices) {
      return;
      //shutdown_wstacking(NO_ACCELERATORS_FOUND, "No accelerators found", __FILE__, __LINE__ );
    }
  }

  hipError_t mmm;
  hipfftResult_t status;

  hipfftDoubleComplex *fftwgrid;

  
  // Alloco fftwgrid su GPU utilizzando hipMalloc

  long long unsigned size_finta_fft = (long long unsigned)((long long unsigned)xaxis*(long long unsigned)yaxis);

  mmm=hipMalloc(&fftwgrid, (size_t)(size_finta_fft*sizeof(hipfftDoubleComplex)));
  if (mmm != hipSuccess) {printf("!!! cuda_fft.cu hipMalloc ERROR %d !!!\n", mmm);}

  int Nth = 32;
  myuint Nbl = (myuint)((yaxis*xaxis)/Nth + 1);  
  

  // Plan creation

  hipfftHandle plan;
  status = hipfftCreate(&plan);
  if (status != HIPFFT_SUCCESS) {printf("!!! hipfftCreate ERROR %d !!!\n", status);}

  hipStream_t stream{};
  hipStreamCreate(&stream);


  status = cufftMpAttachComm(plan, CUFFT_COMM_MPI, &comm);
  if (status != HIPFFT_SUCCESS) {printf("!!! cufftMpAttachComm ERROR %d !!!\n", status);}

  status = hipfftSetStream(plan, stream);
  if (status != HIPFFT_SUCCESS) {printf("!!! hipfftSetStream ERROR %d !!!\n", status);}

  size_t workspace;
  status = hipfftMakePlan2d(plan, grid_size_x, grid_size_y, HIPFFT_Z2Z, &workspace);
  if (status != HIPFFT_SUCCESS) {printf("!!! hipfftMakePlan2d ERROR %d !!!\n", status);}
  hipDeviceSynchronize();


  double norm = 1.0/(double)(grid_size_x*grid_size_y);
  
  // Grid composition
  hipLibXtDesc *fftwgrid_g;
  hipLibXtDesc *fftwgrid_g2;

  
  status = hipfftXtMalloc(plan, &fftwgrid_g2, HIPFFT_XT_FORMAT_INPLACE);
  if (status != HIPFFT_SUCCESS) {printf("!!! hipfftXtMalloc 2 ERROR %d !!!\n", status);}
  hipDeviceSynchronize();
  
  
  mmm = hipStreamSynchronize(stream);
  if (mmm != hipSuccess) {printf("!!! hipStreamSynchronize ERROR %d !!!\n", mmm);}

  for (int iw = 0; iw < num_w_planes; iw++)
    {
        
      //printf("Task %d, FFTing plane %d...\n", rank, iw);

      //Define fftwgrid with a cuda kernel
      write_grid<<<Nbl, Nth>>>(num_w_planes, xaxis, yaxis, fftwgrid, grid, iw);
      hipDeviceSynchronize();

      //Allocate the first descriptor inside the loop
      status = hipfftXtMalloc(plan, &fftwgrid_g, HIPFFT_XT_FORMAT_INPLACE);
      if (status != HIPFFT_SUCCESS) {printf("!!! hipfftXtMalloc ERROR %d !!!\n", status);}

      hipStreamSynchronize(stream);

      //Copy the array to be transformed onto the descriptor structure array 
      mmm = hipMemcpy(fftwgrid_g->descriptor->data[0], fftwgrid, (size_t)(size_finta_fft*sizeof(hipfftDoubleComplex)), hipMemcpyDeviceToDevice);
      if (mmm != hipSuccess) {printf("!!! hipMemcpy 1 ERROR %d !!!\n", mmm);}

      //Perform the FFT
      status = hipfftXtExecDescriptor(plan, fftwgrid_g, fftwgrid_g, HIPFFT_BACKWARD);
      if (status != HIPFFT_SUCCESS) {printf("!!! hipfftXtExecDescriptor ERROR %d !!!\n", status);}

      mmm = hipStreamSynchronize(stream);
      if (mmm != hipSuccess) {printf("!!! hipStreamSynchronize 2 ERROR %d !!!\n", mmm);}

      hipDeviceSynchronize();

      //Put the data in the correct order as required by cufftMP
      status = hipfftXtMemcpy(plan, fftwgrid_g2, fftwgrid_g, HIPFFT_COPY_DEVICE_TO_DEVICE);
      if (status != HIPFFT_SUCCESS) {printf("!!! hipfftXtMemcpy dtd fftwgrid ERROR %d !!!\n", status);}

      //Copy the result descriptor structure array again onto the original fftwgrid
      mmm = hipMemcpy(fftwgrid, fftwgrid_g2->descriptor->data[0], (size_t)(size_finta_fft*sizeof(hipfftDoubleComplex)), hipMemcpyDeviceToDevice);
      if (mmm != hipSuccess) {printf("!!! hipMemcpy 2 ERROR %d !!!\n", mmm);}

      //Write gridss starting from fftwgrid
      write_gridss<<<Nbl, Nth>>>(num_w_planes, xaxis, yaxis, fftwgrid, gridss, norm, iw);

      //Free the first descriptor
      status=hipfftXtFree(fftwgrid_g);
      if (status != HIPFFT_SUCCESS) {printf("!!! hipFree fftwgrid_g ERROR %d !!!\n", mmm);}

      hipDeviceSynchronize();
      
    }

  status=hipfftXtFree(fftwgrid_g2);
  if (status != HIPFFT_SUCCESS) {printf("!!! hipFree fftwgrid_g2 ERROR %d !!!\n", mmm);}
  status = hipfftDestroy(plan);
  if (status != HIPFFT_SUCCESS) {printf("!!! hipfftDestroy fftwgrid ERROR %d !!!\n", status);}
  
  mmm = hipFree(grid);
  if (mmm != hipSuccess) {printf("!!! hipFree grid ERROR %d !!!\n", mmm);}
  
  mmm = hipFree(fftwgrid);
  if (mmm != hipSuccess) {printf("!!! hipFree fftwgrid ERROR %d !!!\n", mmm);}

  hipStreamDestroy(stream);

}
#endif
