#include "hip/hip_runtime.h"
#ifdef _OPENMP
#include <omp.h>
#endif
#include "w-stacking.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

#ifdef __HIPCC__
#include "allvars_nccl.h"
#elif HIPCC
#include "allvars_rccl.h"
#else
#include "allvars.h"
#endif

#include "proto.h"

#ifdef ACCOMP
#pragma omp  declare target
#endif
#ifdef __HIPCC__
double __device__
#else
double
#endif
// Gaussian Kernel
gauss_kernel_norm(double norm, double std22, double u_dist, double v_dist)
{
     double conv_weight;
     conv_weight = norm * exp(-((u_dist*u_dist)+(v_dist*v_dist))*std22);
     return conv_weight;
}

void makeGaussKernel(double * kernel,
		     int KernelLen,
		     int increaseprecision,
		     double std22)
{

  double norm = std22/PI;
  int n = increaseprecision*KernelLen, mid = n / 2;
  for (int i = 0; i != mid + 1; i++) {
      double term = (double)i/(double)increaseprecision;
      kernel[mid + i] = sqrt(norm) * exp(-(term*term)*std22);
  }

  for (int i = 0; i != mid; i++) kernel[i] = kernel[n - 1 - i];
//  for (int i = 0; i < n; i++) printf("%f\n",kernel[i]);

}

// Kaiser-Bessel Kernel: it is adapted from WSClean
double bessel0(double x, double precision) {
  // Calculate I_0 = SUM of m 0 -> inf [ (x/2)^(2m) ]
  // This is the unnormalized bessel function of order 0.
  double d = 0.0, ds = 1.0, sum = 1.0;
  do {
    d += 2.0;
    ds *= x * x / (d * d);
    sum += ds;
  } while (ds > sum * precision);
  return sum;
}
void makeKaiserBesselKernel(double * kernel,
		            int KernelLen,
			    int increaseprecision,
                            double alpha,
                            double overSamplingFactor,
                            int withSinc) {
  int n = increaseprecision*KernelLen, mid = n / 2;
  double * sincKernel = (double*)malloc((mid + 1) * sizeof(*sincKernel));
  const double filterRatio = 1.0 / overSamplingFactor;
  sincKernel[0] = filterRatio;
  for (int i = 1; i != mid + 1; i++) {
    double x = i;
    sincKernel[i] =
        withSinc ? (sin(PI * filterRatio * x) / (PI * x)) : filterRatio;
  }
  const double normFactor = overSamplingFactor / bessel0(alpha, 1e-8);
  for (int i = 0; i != mid + 1; i++) {
    double term = (double)i / mid;
    kernel[mid + i] = sincKernel[i] *
                bessel0(alpha * sqrt(1.0 - (term * term)), 1e-8) *
                normFactor;
  }
  for (int i = 0; i != mid; i++) kernel[i] = kernel[n - 1 - i];
  //for (int i = 0; i < n; i++) printf("%f\n",kernel[i]);
}


#ifdef ACCOMP
#pragma omp end declare target
#endif

#ifdef __HIPCC__
//double __device__ gauss_kernel_norm(double norm, double std22, double u_dist, double v_dist)
//{
//     double conv_weight;
//     conv_weight = norm * exp(-((u_dist*u_dist)+(v_dist*v_dist))*std22);
//     return conv_weight;
//}

__global__ void convolve_g(
			   int num_w_planes,
			   myuint num_points,
			   myuint freq_per_chan,
			   myuint polarizations,
			   double* uu,
			   double* vv,
			   double* ww,
			   float* vis_real,
			   float* vis_img,
			   float* weight,
			   double dx,
			   double dw,
			   int KernelLen,
			   int grid_size_x,
			   int grid_size_y,
			   double* grid,
			  #if defined(GAUSS_HI_PRECISION)
			   double std22
			  #else
			   double std22,
			   double* convkernel
			  #endif
			   )
			   


{
  //printf("DENTRO AL KERNEL\n");
  myuint gid = blockIdx.x*blockDim.x + threadIdx.x;
  if(gid < num_points)
    {
      myuint i = gid;
      myull visindex = i*freq_per_chan*polarizations;
      double norm = std22/PI;

      int j, k;

      /* Convert UV coordinates to grid coordinates. */
      double pos_u = uu[i] / dx;
      double pos_v = vv[i] / dx;
      double ww_i  = ww[i] / dw;

      int grid_w = (int)ww_i;
      int grid_u = (int)pos_u;
      int grid_v = (int)pos_v;

      // check the boundaries
      myuint jmin = (grid_u > KernelLen - 1) ? grid_u - KernelLen : 0;
      myuint jmax = (grid_u < grid_size_x - KernelLen) ? grid_u + KernelLen : grid_size_x - 1;
      myuint kmin = (grid_v > KernelLen - 1) ? grid_v - KernelLen : 0;
      myuint kmax = (grid_v < grid_size_y - KernelLen) ? grid_v + KernelLen : grid_size_y - 1;


      // Convolve this point onto the grid.
      for (k = kmin; k <= kmax; k++)
        {

	  double v_dist = (double)k+0.5 - pos_v;
	  int increaseprecision = 5;
	  
	  for (j = jmin; j <= jmax; j++)
            {
	      double u_dist = (double)j+0.5 - pos_u;
	      myuint iKer = 2 * (j + k*grid_size_x + grid_w*grid_size_x*grid_size_y);
	      int jKer = (int)(increaseprecision * (fabs(u_dist+(double)KernelLen)));
	      int kKer = (int)(increaseprecision * (fabs(v_dist+(double)KernelLen)));
	      
	     #ifdef GAUSS_HI_PRECISION
	      double conv_weight = gauss_kernel_norm(norm,std22,u_dist,v_dist);
	     #endif
	     #ifdef GAUSS
	      double conv_weight = convkernel[jKer]*convkernel[kKer];
	     #endif
	     #ifdef KAISERBESSEL
	      double conv_weight = convkernel[jKer]*convkernel[kKer];
	     #endif

	      // Loops over frequencies and polarizations
	      double add_term_real = 0.0;
	      double add_term_img = 0.0;
	      myull ifine = visindex;
	      for (myuint ifreq=0; ifreq<freq_per_chan; ifreq++)
		{
		  myuint iweight = visindex/freq_per_chan;
		  for (myuint ipol=0; ipol<polarizations; ipol++)
		    {
                      double vistest = (double)vis_real[ifine];
                      if (!isnan(vistest))
			{
			  add_term_real += weight[iweight] * vis_real[ifine] * conv_weight;
			  add_term_img += weight[iweight] * vis_img[ifine] * conv_weight;
			}
                      ifine++;
		      iweight++;
		    }
		}
	      atomicAdd(&(grid[iKer]),add_term_real);
	      atomicAdd(&(grid[iKer+1]),add_term_img);
            }
        }
    }
}
#endif
#ifdef ACCOMP
#pragma  omp declare target
#endif
void wstack(
#ifdef __HIPCC__
	    long long unsigned num_w_planes,
#else
	    int num_w_planes,
#endif
	    myuint num_points,
	    myuint freq_per_chan,
	    myuint polarizations,
	    double* uu,
	    double* vv,
	    double* ww,
	    float* vis_real,
	    float* vis_img,
	    float* weight,
	    double dx,
	    double dw,
	    int w_support,
#ifdef __HIPCC__
	    long long unsigned grid_size_x,
	    long long unsigned grid_size_y,
#else
	    int grid_size_x,
	    int grid_size_y,
#endif
	    double* grid,
	    int num_threads,
#ifdef NCCL_REDUCE
	    int rank,
	    hipStream_t stream_stacking
#else
	    int rank
#endif
            )
{
  myuint i;
  //myuint index;
  myull visindex;
  
  // initialize the convolution kernel
  // gaussian:
  int KernelLen = (w_support-1)/2;
  int increaseprecision = 5; // this number must be odd: increaseprecison*w_support must be odd (w_support must be odd)
  double std = 1.0;
  double std22 = 1.0/(2.0*std*std);
  double norm = std22/PI;
  double * convkernel = (double*)malloc(increaseprecision*w_support*sizeof(*convkernel));
  
#ifdef GAUSS
  makeGaussKernel(convkernel,w_support,increaseprecision,std22);
#endif
#ifdef KAISERBESSEL
  double overSamplingFactor = 1.0;
  int withSinc = 0;
  double alpha = 8.6;
  makeKaiserBesselKernel(convkernel, w_support, increaseprecision, alpha, overSamplingFactor, withSinc);
#endif
  
  
  // Loop over visibilities.
  // Switch between CUDA and GPU versions
#ifdef __HIPCC__
  // Define the CUDA set up
  int Nth = NTHREADS;
  myuint Nbl = (myuint)(num_points/Nth) + 1;
  if(NWORKERS == 1) {Nbl = 1; Nth = 1;};
  myull Nvis = num_points*freq_per_chan*polarizations;
  
  int ndevices;
  hipGetDeviceCount(&ndevices);
  hipSetDevice(rank % ndevices);
  
  if ( rank == 0 ) {
    if (0 == ndevices) {
      
      shutdown_wstacking(NO_ACCELERATORS_FOUND, "No accelerators found", __FILE__, __LINE__ );
    }
  }
  
#ifdef NVIDIA
  prtAccelInfo();
#endif
  
  // Create GPU arrays and offload them
  double * uu_g;
  double * vv_g;
  double * ww_g;
  float * vis_real_g;
  float * vis_img_g;
  float * weight_g;
  double * convkernel_g;
#if !defined(NCCL_REDUCE)
  double * grid_g;
#endif
#if !defined(NCCL_REDUCE)
  hipStream_t stream_stacking;
  hipStreamCreate(&stream_stacking);
#endif
  
  //Create the event inside stream stacking
  //hipEvent_t event_kernel;
  
  //for (int i=0; i<100000; i++)grid[i]=23.0;
  hipError_t mmm;
  //mmm=hipEventCreate(&event_kernel);
  mmm=hipMalloc(&uu_g,num_points*sizeof(double));
  mmm=hipMalloc(&vv_g,num_points*sizeof(double));
  mmm=hipMalloc(&ww_g,num_points*sizeof(double));
  mmm=hipMalloc(&vis_real_g,Nvis*sizeof(float));
  mmm=hipMalloc(&vis_img_g,Nvis*sizeof(float));
  mmm=hipMalloc(&weight_g,(Nvis/freq_per_chan)*sizeof(float));
  //mmm=hipMalloc(&grid_g,2*num_w_planes*grid_size_x*grid_size_y*sizeof(double));

#if !defined(NCCL_REDUCE)
  mmm = hipMalloc(&grid_g,2*num_w_planes*grid_size_x*grid_size_y*sizeof(double));
#endif

#if !defined(GAUSS_HI_PRECISION)
  mmm=hipMalloc(&convkernel_g,increaseprecision*w_support*sizeof(double));
#endif    
  if (mmm != hipSuccess) {printf("!!! w-stacking.cu hipMalloc ERROR %d !!!\n", mmm);}
    
#if !defined(NCCL_REDUCE)
  mmm=hipMemset(grid_g,0.0,2*num_w_planes*grid_size_x*grid_size_y*sizeof(double));
  if (mmm != hipSuccess) {printf("!!! w-stacking.cu hipMemset ERROR %d !!!\n", mmm);}
#endif
    
  mmm=hipMemcpyAsync(uu_g, uu, num_points*sizeof(double), hipMemcpyHostToDevice, stream_stacking);
  mmm=hipMemcpyAsync(vv_g, vv, num_points*sizeof(double), hipMemcpyHostToDevice, stream_stacking);
  mmm=hipMemcpyAsync(ww_g, ww, num_points*sizeof(double), hipMemcpyHostToDevice, stream_stacking);
  mmm=hipMemcpyAsync(vis_real_g, vis_real, Nvis*sizeof(float), hipMemcpyHostToDevice, stream_stacking);
  mmm=hipMemcpyAsync(vis_img_g, vis_img, Nvis*sizeof(float), hipMemcpyHostToDevice, stream_stacking);
  mmm=hipMemcpyAsync(weight_g, weight, (Nvis/freq_per_chan)*sizeof(float), hipMemcpyHostToDevice, stream_stacking);


#if !defined(GAUSS_HI_PRECISION)
  mmm=hipMemcpyAsync(convkernel_g, convkernel, increaseprecision*w_support*sizeof(double), hipMemcpyHostToDevice, stream_stacking);
#endif

  if (mmm != hipSuccess) {printf("!!! w-stacking.cu hipMemcpyAsync ERROR %d !!!\n", mmm);}

  // Call main GPU Kernel
#if defined(GAUSS_HI_PRECISION)
  convolve_g <<<Nbl,Nth,0,stream_stacking>>> (
					      num_w_planes,
					      num_points,
					      freq_per_chan,
					      polarizations,
					      uu_g,
					      vv_g,
					      ww_g,
					      vis_real_g,
					      vis_img_g,
					      weight_g,
					      dx,
					      dw,
					      KernelLen,
					      grid_size_x,
					      grid_size_y,
#if !defined(NCCL_REDUCE)
					      grid_g,
#else
					      grid,
#endif
					      std22
					      );
#else
  convolve_g <<<Nbl,Nth,0,stream_stacking>>> (
					      num_w_planes,
					      num_points,
					      freq_per_chan,
					      polarizations,
					      uu_g,
					      vv_g,
					      ww_g,
					      vis_real_g,
					      vis_img_g,
					      weight_g,
					      dx,
					      dw,
					      KernelLen,
					      grid_size_x,
					      grid_size_y,
#if !defined(NCCL_REDUCE)
					      grid_g,
#else
					      grid,
#endif
					      std22,
					      convkernel_g
					      );
#endif    
    
  mmm=hipStreamSynchronize(stream_stacking);
  //Record the event
  //mmm=hipEventRecord(event_kernel,stream_stacking);
    
  //Wait until the kernel ends
  //mmm=hipStreamWaitEvent(stream_stacking,event_kernel);
    
  //for (int i=0; i<100000; i++)printf("%f\n",grid[i]);
    
#if !defined(NCCL_REDUCE)
  mmm=hipMemcpy(grid, grid_g, 2*num_w_planes*grid_size_x*grid_size_y*sizeof(double), hipMemcpyDeviceToHost);
#endif

  if (mmm != hipSuccess)
    printf("CUDA ERROR %s\n",hipGetErrorString(mmm));
    
  mmm=hipFree(uu_g);
  mmm=hipFree(vv_g);
  mmm=hipFree(ww_g);
  mmm=hipFree(vis_real_g);
  mmm=hipFree(vis_img_g);
  mmm=hipFree(weight_g);

#if !defined(NCCL_REDUCE)
  mmm=hipFree(grid_g);
#endif
    
#if !defined(GAUSS_HI_PRECISION)
  mmm=hipFree(convkernel_g);
#endif

  // Switch between CUDA and GPU versions
# else

#ifdef _OPENMP
  omp_set_num_threads(num_threads);
#endif

#if defined(ACCOMP) && (GPU_STACKING)
  omp_set_default_device(rank % omp_get_num_devices());
  myull Nvis = num_points*freq_per_chan*polarizations;
#pragma omp target teams distribute parallel for private(visindex) map(to:uu[0:num_points], vv[0:num_points], ww[0:num_points], vis_real[0:Nvis], vis_img[0:Nvis], weight[0:Nvis/freq_per_chan]) map(tofrom:grid[0:2*num_w_planes*grid_size_x*grid_size_y])
#else
#pragma omp parallel for private(visindex)
#endif
  for (i = 0; i < num_points; i++)
    {
#ifdef _OPENMP
      //int tid;
      //tid = omp_get_thread_num();
      //printf("%d\n",tid);
#endif

      visindex = i*freq_per_chan*polarizations;

      double sum = 0.0;
      int j, k;
      //if (i%1000 == 0)printf("%ld\n",i);

      /* Convert UV coordinates to grid coordinates. */
      double pos_u = uu[i] / dx;
      double pos_v = vv[i] / dx;
      double ww_i  = ww[i] / dw;
	
      int grid_w = (int)ww_i;
      int grid_u = (int)pos_u;
      int grid_v = (int)pos_v;

      // check the boundaries
      myuint jmin = (grid_u > KernelLen - 1) ? grid_u - KernelLen : 0;
      myuint jmax = (grid_u < grid_size_x - KernelLen) ? grid_u + KernelLen : grid_size_x - 1;
      myuint kmin = (grid_v > KernelLen - 1) ? grid_v - KernelLen : 0;
      myuint kmax = (grid_v < grid_size_y - KernelLen) ? grid_v + KernelLen : grid_size_y - 1;
      //printf("%d, %ld, %ld, %d, %ld, %ld\n",grid_u,jmin,jmax,grid_v,kmin,kmax);


      // Convolve this point onto the grid.
      for (k = kmin; k <= kmax; k++)
        {

	  double v_dist = (double)k+0.5 - pos_v;
	  //double v_dist = (double)k - pos_v;

	  for (j = jmin; j <= jmax; j++)
            {
	      double u_dist = (double)j+0.5 - pos_u;
	      //double u_dist = (double)j - pos_u;
	      myuint iKer = 2 * (j + k*grid_size_x + grid_w*grid_size_x*grid_size_y);
	      int jKer = (int)(increaseprecision * (fabs(u_dist+(double)KernelLen)));
	      int kKer = (int)(increaseprecision * (fabs(v_dist+(double)KernelLen)));

#ifdef GAUSS_HI_PRECISION
	      double conv_weight = gauss_kernel_norm(norm,std22,u_dist,v_dist);
#endif
#ifdef GAUSS
	      double conv_weight = convkernel[jKer]*convkernel[kKer];
	      //if(jKer < 0 || jKer >= 35 || kKer < 0 || kKer >= 35)
	      //	printf("%f %d %f %d\n",fabs(u_dist+(double)KernelLen),jKer,fabs(v_dist+(double)KernelLen),kKer);
	      //printf("%d %d %d %d %f %f %f %f %f\n",jKer, j, kKer, k, pos_u, pos_v, u_dist,v_dist,conv_weight);
#endif
#ifdef KAISERBESSEL
	      double conv_weight = convkernel[jKer]*convkernel[kKer];
#endif
	      // Loops over frequencies and polarizations
	      double add_term_real = 0.0;
	      double add_term_img = 0.0;
	      myull ifine = visindex;
	      // DAV: the following two loops are performend by each thread separately: no problems of race conditions
	      for (myuint ifreq=0; ifreq<freq_per_chan; ifreq++)
		{
		  myuint iweight = visindex/freq_per_chan;
		  for (myuint ipol=0; ipol<polarizations; ipol++)
		    {
                      if (!isnan(vis_real[ifine]))
			{
			  //printf("%f %ld\n",weight[iweight],iweight);
			  add_term_real += weight[iweight] * vis_real[ifine] * conv_weight;
			  add_term_img += weight[iweight] * vis_img[ifine] * conv_weight;
			  //if(vis_img[ifine]>1e10 || vis_img[ifine]<-1e10)printf("%f %f %f %f %ld %ld\n",vis_real[ifine],vis_img[ifine],weight[iweight],conv_weight,ifine,num_points*freq_per_chan*polarizations);
			}
		      ifine++;
		      iweight++;
		    }
	        }
	      // DAV: this is the critical call in terms of correctness of the results and of performance
#pragma omp atomic
	      grid[iKer] += add_term_real;
#pragma omp atomic
	      grid[iKer+1] += add_term_img;
            }
        }
	
    }
#if defined(ACCOMP) && (GPU_STACKING)
#pragma omp target exit data map(delete:uu[0:num_points], vv[0:num_points], ww[0:num_points], vis_real[0:Nvis], vis_img[0:Nvis], weight[0:Nvis/freq_per_chan], grid[0:2*num_w_planes*grid_size_x*grid_size_y])
#endif
  // End switch between CUDA and CPU versions
#endif
  //for (int i=0; i<100000; i++)printf("%f\n",grid[i]);
}

#ifdef ACCOMP
#pragma  omp end declare target
#endif

int test(int nnn)
{
  int mmm;

  mmm = nnn+1;
  return mmm;
}
